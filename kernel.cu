

#include <hip/hip_runtime.h>
#include <stdio.h>


extern "C" __global__ void kernel ( int cnt, int* buf  )
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	if ( x > cnt ) return;

	buf[x] = 1000-x;

}
