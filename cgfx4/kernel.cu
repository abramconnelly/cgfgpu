#include "hip/hip_runtime.h"

#include <stdio.h>

#include "cuda_math.cuh"

__constant__ int tab32[32] = {
	0,   9, 1,  10, 13, 21,  2, 29,
	11, 14, 16, 18, 22, 25,  3, 30,
	8,  12, 20, 28, 15, 17, 24,  7,
	19, 27, 23,  6, 26,  5,  4, 31 };

__device__ inline int log2_32(uint32_t value) {
	value |= value >> 1;
	value |= value >> 2;
	value |= value >> 4;
	value |= value >> 8;
	value |= value >> 16;
	return tab32[(uint32_t)(value * 0x07C4ACDD) >> 27];
}

__device__ inline int NumberOfSetBits32(uint32_t u)
{
  u = u - ((u >> 1) & 0x55555555);
  u = (u & 0x33333333) + ((u >> 2) & 0x33333333);
  return (((u + (u >> 4)) & 0x0F0F0F0F) * 0x01010101) >> 24;
}


extern "C" __global__ void concordanceKernel ( 
			int cgfa, 
			int blk_start, int blk_end,
			int mStartPath, int mStartStep, int mEndPath, int mEndStep, int mStartTile,
			int mTotalBlocks, int mTotalGenomes,
			uint32_t* mCanonVec, uint32_t* mCacheVec, uint32_t* mSpanVec, uint32_t* mLoqVec,
			uchar* mMatchList, uchar* mTotalList, uint8_t* mTable )
{
	int ii = blockIdx.x * blockDim.x + threadIdx.x + blk_start;		// block id
	int cgfb = blockIdx.y * blockDim.y + threadIdx.y;				// second genome
	if ( ii < blk_start || ii > blk_end ) return;
	if ( cgfb > mTotalGenomes ) return;

	// input reads
	uint32_t hiq_a, hiq_b;
	uint32_t span_a, span_b;	
	uint32_t canon_a, canon_b;	
	uint32_t cache_a, cache_b;
	
	// computed
	uint32_t actual_canon_a, actual_canon_b;
	uint32_t anchor_mask_a, anchor_mask_b;
	uint32_t non_anchor_span_mask_a, non_anchor_span_mask_b;
	uint32_t tmp;

	int hexit_a[8], hexit_b[8],
		hexit_relative_step_a[8], hexit_relative_step_b[8];

	int i, j, p, k;

	// Input bit vector s
	hiq_a = ~ *( (mLoqVec   + cgfa*mTotalBlocks) + ii);		// flip loq_a
	hiq_b = ~ *( (mLoqVec   + cgfb*mTotalBlocks) + ii);		// flip loq_b
	span_a =  *( (mSpanVec  + cgfa*mTotalBlocks) + ii);		// span bits
	span_b =  *( (mSpanVec  + cgfb*mTotalBlocks) + ii);	
	canon_a = *( (mCanonVec + cgfa*mTotalBlocks) + ii);		// canon bits
	canon_b = *( (mCanonVec + cgfb*mTotalBlocks) + ii);
	cache_a = *( (mCacheVec + cgfa*mTotalBlocks) + ii);		// cache bits
	cache_b = *( (mCacheVec + cgfb*mTotalBlocks) + ii);
	
	// Computed bit-vectors
	actual_canon_a = canon_a & ~span_a & hiq_a;		// get actual canon bits, masking off the span bit rules
	anchor_mask_a = span_a & hiq_a & (~canon_a);	// anchor tile bit vector for convenience.
	non_anchor_span_mask_a = span_a & (~anchor_mask_a);

	actual_canon_b = canon_b & ~span_b & hiq_b;
	anchor_mask_b = span_b & hiq_b & (~canon_b);
	non_anchor_span_mask_b = span_b & (~anchor_mask_b);

	// Cache values for genome a, pulled out into array 	
	hexit_a[0] = (cache_a & ((uint32_t)0xf));
	hexit_a[1] = (cache_a & ((uint32_t)0xf << 4)) >> 4;
	hexit_a[2] = (cache_a & ((uint32_t)0xf << 8)) >> 8;
	hexit_a[3] = (cache_a & ((uint32_t)0xf << 12)) >> 12;
	hexit_a[4] = (cache_a & ((uint32_t)0xf << 16)) >> 16;
	hexit_a[5] = (cache_a & ((uint32_t)0xf << 20)) >> 20;
	hexit_a[6] = (cache_a & ((uint32_t)0xf << 24)) >> 24;
	hexit_a[7] = (cache_a & ((uint32_t)0xf << 28)) >> 28;
	
	// Fill out the hexit_relative_step array
	// This is a count of the canonical bits to get the cache entry positions.
	//
	tmp = (anchor_mask_a & hiq_a) | ((~span_a) & (~actual_canon_a) & hiq_a);	// overflow_a: non-canonical tiles and anchor tiles, excluding spanning tiles.
	for (k = 0; (k<8) && (tmp != 0); k++, tmp &= (tmp - 1)) {
		hexit_relative_step_a[k] = log2_32(tmp & ~(tmp - 1));
	}

	// Cache values for genome b, pulled out into array 
	//
	hexit_b[0] = (cache_b & ((uint32_t)0xf));
	hexit_b[1] = (cache_b & ((uint32_t)0xf << 4)) >> 4;
	hexit_b[2] = (cache_b & ((uint32_t)0xf << 8)) >> 8;
	hexit_b[3] = (cache_b & ((uint32_t)0xf << 12)) >> 12;
	hexit_b[4] = (cache_b & ((uint32_t)0xf << 16)) >> 16;
	hexit_b[5] = (cache_b & ((uint32_t)0xf << 20)) >> 20;
	hexit_b[6] = (cache_b & ((uint32_t)0xf << 24)) >> 24;
	hexit_b[7] = (cache_b & ((uint32_t)0xf << 28)) >> 28;

	// Fill out the hexit_relative_step array
	// This is a count of the canonical bits to get the cache entry positions.
	//
	tmp = (anchor_mask_b & hiq_b) | ((~span_b) & (~actual_canon_b) & hiq_b);	// overflow_b
	for (k = 0; (k<8) && (tmp != 0); k++, tmp &= (tmp - 1)) {
		hexit_relative_step_b[k] = log2_32(tmp & ~(tmp - 1));
	}
	// Count the number of canonical matches.
	tmp = NumberOfSetBits32(actual_canon_a & actual_canon_b);

	// Cache value testing
	//
	// Do a zipper match to count the number of cache overflow hits.
	// Tile variants that overflow from the cache will be picked
	// up by the overflow count.
	//
	for (i = 0, j = 0; (i<8) && (j<8); ) {

		if ((hexit_relative_step_a[i] < 0) || (hexit_relative_step_b[j] < 0)) { break; }

		if (hexit_relative_step_a[i] < hexit_relative_step_b[j]) {
			i++; continue;
		}
		if (hexit_relative_step_a[i] > hexit_relative_step_b[j]) {
			j++; continue;
		}
		if (hexit_relative_step_a[i] == hexit_relative_step_b[j]) {
			if ((hexit_a[i] > 0) && (hexit_a[i] < 0xf) &&
				(hexit_b[j] > 0) && (hexit_b[j] < 0xf) &&
				(hexit_a[i] == hexit_b[j])) {

				if ((ii > blk_start) && (ii < blk_end)) {
					tmp++;
				}

				else if ((ii == blk_start) && (ii < blk_end) &&
					((mStartTile + hexit_relative_step_a[i]) >= mStartStep) &&
					((mStartTile + hexit_relative_step_b[j]) >= mStartStep)) {
					tmp++;
				}

				else if ((ii > blk_start) && (ii == blk_end) &&
					((mStartTile + hexit_relative_step_a[i]) <= mEndStep) &&
					((mStartTile + hexit_relative_step_b[j]) <= mEndStep)) {
					tmp++;
				}

				else if ((ii == blk_start) && (ii == blk_end) &&
					((mStartTile + hexit_relative_step_a[i]) >= mStartStep) &&
					((mStartTile + hexit_relative_step_b[j]) >= mStartStep) &&
					((mStartTile + hexit_relative_step_a[i]) <= mEndStep) &&
					((mStartTile + hexit_relative_step_b[j]) <= mEndStep)) {
					tmp++;
				}
				// else we skip over and don't count the match as it falls outside of the selected window
			}
			i++;
			j++;
		}
	}


	// The total number of high quality matches is the number of high quality
	// tiles both have in common, excluding the non-anchor high quality tiles
	//
	* (mMatchList + (cgfb*mTotalBlocks) + ii) = tmp;
	* (mTotalList + (cgfb*mTotalBlocks) + ii) = NumberOfSetBits32(hiq_a & hiq_b & (~non_anchor_span_mask_a) & (~non_anchor_span_mask_b));

}


